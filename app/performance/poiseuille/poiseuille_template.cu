#include <iostream>
#include <vector>
#include <utility>
#include <stdexcept>
#include <fstream>

#include <boost/static_assert.hpp>
#include <boost/type_traits.hpp>

// thrust includes
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/binary_search.h>

// fusion includes
#include <boost/fusion/sequence.hpp>
#include <boost/fusion/algorithm.hpp>
#include <boost/fusion/container.hpp>

// cuda include
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <omp.h>

// algebra includes
#include <boost/numeric/odeint/external/thrust/thrust_algebra.hpp>

#include <mpc/defines.hpp>
#include <mpc/point.hpp>
#include <mpc/mass_point.hpp>
#include <mpc/boundaries.hpp>
#include <mpc/streaming.hpp>
#include <mpc/indexer.hpp>
#include <mpc/collision.hpp>
#include <mpc/analysis.hpp>
#include <mpc/drand48_generator.hpp>

using namespace std;
using namespace mpc2;
namespace fusion = boost::fusion;

using boost::numeric::odeint::thrust_algebra;


const static size_t dim = 2;
typedef double value_type;
typedef point< value_type , dim > point_type;
typedef mass_point< point_type > mass_point_type;

typedef VECTOR< mass_point_type > mass_point_vector_type;
typedef VECTOR< point_type > point_vector_type;
typedef VECTOR< size_t > cell_type;
typedef VECTOR< point< size_t , dim > > index_type;
typedef VECTOR< hiprandState > rng_state_type;


const point_type null_point( 0.0 , 0.0 );
const point_type spacing( 1.0 , 1.0 );

RANGE
DENSITY


// const point_type range( 15.0 , 8.0 );
// const value_type density = 5;
const size_t n_solvent = size_t( range[0] * range[1] / spacing[0] / spacing[1] * density );
const size_t n_virtual_particles = range[0] / spacing[0] * density;
const size_t n = n_solvent + n_virtual_particles;

const point_type gravitation( 0.001 , 0.0 );


const value_type sqrt_kbT_mass = 1.0;
const value_type mass = 1.0;
const value_type dt = 0.025;




template
<
	class MassPointVector ,
	class PointVector ,
	class BC ,
	class Indexer ,
	class IndexVector ,
	class Rng ,
	class Value
>
void test_poiseuille
(
	MassPointVector &mp ,
	PointVector &force ,
	size_t num_of_particles , size_t num_of_virtual_particles ,
	BC &bc ,
	Indexer &indexer ,
	IndexVector &indices , IndexVector &permutated_indices ,
	IndexVector &begins ,
	IndexVector &ends ,
	Rng &rng ,
	Value &t , Value dt , Value mass ,
	size_t num_of_streaming_steps
)
{
	typedef streaming< thrust_algebra , cuda_streaming_operations > streaming_type;

	size_t num_of_all_particles = mp.size() , num_of_cells = indexer.number_of_cells();

	if( ( num_of_all_particles != ( num_of_particles + num_of_virtual_particles ) ) ||
		( force.size() != num_of_all_particles ) ||
		( indices.size() != num_of_all_particles ) ||
		( begins.size() != num_of_cells ) ||
		( ends.size() != num_of_cells ) ||
		( rng.size() != num_of_cells ) )
	{
		throw std::invalid_argument( "Sizes does not match!" );
	}

	for( size_t i=0 ; i<num_of_streaming_steps ; ++i , t+=dt )
	{
		streaming_type::step_a( std::make_pair( mp.begin() , mp.begin() + num_of_particles ) , force , t , dt , mass , bc );
		streaming_type::step_b( std::make_pair( mp.begin() , mp.begin() + num_of_particles ) , force , dt , mass );
	}


	thrust::counting_iterator< size_t > search_begin( 0 );
	thrust::copy( search_begin , search_begin + num_of_all_particles , permutated_indices.begin() );
	thrust::transform( mp.begin() , mp.end() , indices.begin() , make_indexer_caller( indexer ) );
	thrust::sort_by_key( indices.begin() , indices.end() , permutated_indices.begin() );
	thrust::lower_bound( indices.begin() , indices.end() , search_begin , search_begin + num_of_cells , begins.begin() );
	thrust::upper_bound( indices.begin() , indices.end() , search_begin , search_begin + num_of_cells , ends.begin() );
	thrust::for_each(
			thrust::make_zip_iterator( thrust::make_tuple( begins.begin() , ends.begin() , rng.begin() ) ) ,
			thrust::make_zip_iterator( thrust::make_tuple( begins.end() , ends.end() , rng.end() ) ) ,
			anderson_plus_a_permutating< MassPointVector , IndexVector >( mp , permutated_indices )
			);
}







int main( int argc , char **argv )
{
	OMP_NUM_THREADS

	srand48( 325345777 );
	typedef drand48_generator rng_type;
	rng_type rng;


	typedef mpc2::regular_indexer< point_type ,
			with_start_point_tag ,
			check_and_throw_index_policy > regular_indexer_type;
	regular_indexer_type indexer( range , spacing );

	indexer.automatic_grid_shift( drand48 );
//	clog << "Indexer start point : " << indexer.start() << endl;



	// initialize the solvent particles
	std::vector< mass_point_type > tmp_solvent;
	std::vector< point_type > tmp_force;
	for( size_t i=0 ; i<n ; ++i )
	{
		mass_point_type mp;
		mp.coor[0] = drand48() * range[0];
		mp.coor[1] = drand48() * range[1];
		maxwell_velocity( mp.vel , sqrt_kbT_mass , rng );
		tmp_solvent.push_back( mp );
		tmp_force.push_back( gravitation );
	}
	mass_point_vector_type solvent = tmp_solvent;
	point_vector_type force = tmp_force;


	cell_type indices( n ) , permutated_indices( n );
	cell_type begins( indexer.number_of_cells() ) , ends( indexer.number_of_cells() );

	std::vector< hiprandState > tmp_rng_state( indexer.number_of_cells() );
	for( size_t i=0 ; i<indexer.number_of_cells() ; ++i ) hiprand_init( 1856234 , 0 , 0 , &tmp_rng_state[i] );
	rng_state_type rng_state = tmp_rng_state;

	fusion::vector
	<
		mpc2::periodic_boundary< value_type , 0 > ,
		mpc2::straight_bounce_back< point_type , 1 , true > ,
		mpc2::straight_bounce_back< point_type , 1 , false > ,
		mpc2::fixed_boundary_corrector< point_type >
	>
	bc
	(
		mpc2::periodic_boundary< value_type , 0 >( 0.0 , range[0] ) ,
		mpc2::straight_bounce_back< point_type , 1 , true >( 0.0 , dt ) ,
		mpc2::straight_bounce_back< point_type , 1 , false >( range[1] , dt ) ,
		mpc2::fixed_boundary_corrector< point_type >( null_point , range )
	);


	value_type t = 0.0 ;

	for( size_t i=0 ; i<200000 ; ++i )
	{
		test_poiseuille( solvent , force , n_solvent , n_virtual_particles , bc , indexer , indices , permutated_indices , begins , ends , rng_state , t , dt , mass , 4 );
		indexer.automatic_grid_shift( drand48 );
	}

	ofstream fout( OUTFILE );
	thrust::host_vector< mass_point_type > tmp = solvent;
	for( size_t i=0 ; i<tmp.size() ; ++i )
		fout << tmp[i].coor << "\t" << tmp[i].vel << "\n";

	return 0;
}
