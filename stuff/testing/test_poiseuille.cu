/*
 * test_anderson.cu
 *
 *  Created on: Dec 15, 2010
 *      Author: karsten
 */


/*
 * streaming.cpp
 *
 *  Created on: Dec 6, 2010
 *      Author: karsten
 */

#include <iostream>
#include <vector>
#include <utility>
#include <stdexcept>
#include <fstream>

#include <boost/static_assert.hpp>
#include <boost/type_traits.hpp>
#include <boost/random.hpp>

// thrust includes
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/binary_search.h>

// fusion includes
#include <boost/fusion/sequence.hpp>
#include <boost/fusion/algorithm.hpp>
#include <boost/fusion/container.hpp>

// cuda include
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// algebra includes
#include <boost/numeric/odeint/algebra/range_algebra.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_algebra.hpp>

#include <mpc/defines.hpp>
#include <mpc/point.hpp>
#include <mpc/mass_point.hpp>
#include <mpc/boundaries.hpp>
#include <mpc/streaming.hpp>
#include <mpc/indexer.hpp>
#include <mpc/collision.hpp>
#include <mpc/analysis.hpp>

using namespace std;
using namespace mpc2;
namespace fusion = boost::fusion;

using boost::numeric::odeint::range_algebra;
using boost::numeric::odeint::thrust_algebra;


const static size_t dim = 2;
typedef double value_type;
typedef point< value_type , dim > point_type;
typedef mass_point< point_type > mass_point_type;

typedef std::vector< mass_point_type > mass_point_vector_type1;
typedef std::vector< point_type > point_vector_type1;
typedef std::vector< size_t > cell_type1;
typedef std::vector< point< size_t , dim > > index_type1;
typedef std::vector< hiprandState > rng_state_type1;


typedef thrust::host_vector< mass_point_type > mass_point_vector_type2;
typedef thrust::host_vector< point_type > point_vector_type2;
typedef thrust::host_vector< size_t > cell_type2;
typedef thrust::host_vector< point< size_t , dim > > index_type2;
typedef thrust::host_vector< hiprandState > rng_state_type2;


typedef thrust::device_vector< mass_point_type > mass_point_vector_type3;
typedef thrust::device_vector< point_type > point_vector_type3;
typedef thrust::device_vector< size_t > cell_type3;
typedef thrust::device_vector< point< size_t , dim > > index_type3;
typedef thrust::device_vector< hiprandState > rng_state_type3;


const point_type null_point( 0.0 , 0.0 );
const point_type spacing( 1.0 , 1.0 );

const point_type range( 30.0 , 15.0 );
// const point_type range( 15.0 , 8.0 );
const value_type density = 5;
const size_t n_solvent = size_t( range[0] * range[1] / spacing[0] / spacing[1] * density );
const size_t n_virtual_particles = range[0] / spacing[0] * density;
const size_t n = n_solvent + n_virtual_particles;

const point_type gravitation( 0.001 , 0.0 );


const value_type sqrt_kbT_mass = 1.0;
const value_type mass = 1.0;
const value_type dt = 0.025;




template
<
	class MassPointVector ,
	class PointVector ,
	class BC ,
	class Indexer ,
	class IndexVector ,
	class Rng ,
	class Value
>
void test_poiseuille
(
	MassPointVector &mp ,
	PointVector &force ,
	size_t num_of_particles , size_t num_of_virtual_particles ,
	BC &bc ,
	Indexer &indexer ,
	IndexVector &indices , IndexVector &permutated_indices ,
	IndexVector &begins ,
	IndexVector &ends ,
	Rng &rng ,
	Value &t , Value dt , Value mass ,
	size_t num_of_streaming_steps
)
{
	typedef streaming< thrust_algebra , cuda_streaming_operations > streaming_type;

	size_t num_of_all_particles = mp.size() , num_of_cells = indexer.number_of_cells();

	if( ( num_of_all_particles != ( num_of_particles + num_of_virtual_particles ) ) ||
		( force.size() != num_of_all_particles ) ||
		( indices.size() != num_of_all_particles ) ||
		( begins.size() != num_of_cells ) ||
		( ends.size() != num_of_cells ) ||
		( rng.size() != num_of_cells ) )
	{
		throw std::invalid_argument( "Sizes does not match!" );
	}

	for( size_t i=0 ; i<num_of_streaming_steps ; ++i , t+=dt )
	{
		streaming_type::step_a( std::make_pair( mp.begin() , mp.begin() + num_of_particles ) , force , t , dt , mass , bc );
		streaming_type::step_b( std::make_pair( mp.begin() , mp.begin() + num_of_particles ) , force , dt , mass );
//		streaming_type::step_a( mp , force , t , dt , mass , bc );
//		streaming_type::step_b( mp , force , dt , mass );
	}


	thrust::counting_iterator< size_t > search_begin( 0 );
	thrust::copy( search_begin , search_begin + num_of_all_particles , permutated_indices.begin() );

	thrust::transform( mp.begin() , mp.end() , indices.begin() , make_indexer_caller( indexer ) );
	thrust::sort_by_key( indices.begin() , indices.end() , permutated_indices.begin() );
	thrust::lower_bound( indices.begin() , indices.end() , search_begin , search_begin + num_of_cells , begins.begin() );
	thrust::upper_bound( indices.begin() , indices.end() , search_begin , search_begin + num_of_cells , ends.begin() );

	thrust::for_each(
			thrust::make_zip_iterator( thrust::make_tuple( begins.begin() , ends.begin() , rng.begin() ) ) ,
			thrust::make_zip_iterator( thrust::make_tuple( begins.end() , ends.end() , rng.end() ) ) ,
			anderson_plus_a_permutating< MassPointVector , IndexVector >( mp , permutated_indices )
			);
}







int main( int argc , char **argv )
{
	srand48( 325345777 );
	typedef boost::mt19937 rng_type;
	rng_type rng;


	typedef mpc2::regular_indexer< point_type ,
			with_start_point_tag ,
			check_and_throw_index_policy > regular_indexer_type;
	regular_indexer_type indexer( range , spacing );

	indexer.automatic_grid_shift( drand48 );
	clog << "Indexer start point : " << indexer.start() << endl;



	// initialize the solvent particles
	mass_point_vector_type1 solvent1;
	point_vector_type1 force1;
	for( size_t i=0 ; i<n ; ++i )
	{
		mass_point_type mp;
		mp.coor[0] = drand48() * range[0];
		mp.coor[1] = drand48() * range[1];
		maxwell_velocity( mp.vel , sqrt_kbT_mass , rng );
		solvent1.push_back( mp );
		force1.push_back( gravitation );
	}


	mass_point_vector_type2 solvent2 = solvent1;
	mass_point_vector_type3 solvent3 = solvent1;
	point_vector_type2 force2 = force1;
	point_vector_type3 force3 = force1;

	cell_type1 indices1( n ) , permutated_indices1( n );
	cell_type2 indices2( n ) , permutated_indices2( n );
	cell_type3 indices3( n ) , permutated_indices3( n );

	cell_type1 begins1( indexer.number_of_cells() ) , ends1( indexer.number_of_cells() );
	cell_type2 begins2( indexer.number_of_cells() ) , ends2( indexer.number_of_cells() );
	cell_type3 begins3( indexer.number_of_cells() ) , ends3( indexer.number_of_cells() );

	rng_state_type1 rng1( indexer.number_of_cells() );
	for( size_t i=0 ; i<indexer.number_of_cells() ; ++i ) hiprand_init( 1856234 , 0 , 0 , &rng1[i] );
	rng_state_type2 rng2 = rng1;
	rng_state_type3 rng3 = rng1;

	fusion::vector
	<
		mpc2::periodic_boundary< value_type , 0 > ,
		mpc2::straight_bounce_back< point_type , 1 , true > ,
		mpc2::straight_bounce_back< point_type , 1 , false > ,
		mpc2::fixed_boundary_corrector< point_type >
	>
	bc
	(
		mpc2::periodic_boundary< value_type , 0 >( 0.0 , range[0] ) ,
		mpc2::straight_bounce_back< point_type , 1 , true >( 0.0 , dt ) ,
		mpc2::straight_bounce_back< point_type , 1 , false >( range[1] , dt ) ,
		mpc2::fixed_boundary_corrector< point_type >( null_point , range )
	);


	value_type t1 = 0.0 , t2 = 0.0 , t3 = 0.0;
//	test_poiseuille( solvent1 , force1 , n_solvent , n_virtual_particles , bc , indexer , indices1 , permutated_indices1 , begins1 , ends1 , rng1 , t1 , dt , mass , 4 );
//	test_poiseuille( solvent2 , force2 , n_solvent , n_virtual_particles , bc , indexer , indices2 , permutated_indices2 , begins2 , ends2 , rng2 , t2 , dt , mass , 4 );
//	test_poiseuille( solvent3 , force3 , n_solvent , n_virtual_particles , bc , indexer , indices3 , permutated_indices3 , begins3 , ends3 , rng3 , t3 , dt , mass , 4 );

    mpc2::velocity_dist_y< mass_point_type >	vel_dist( indexer.dimensions()[1] , range[1] , 0.0 , range[0] );

//	cout << "set term x11" << endl;
//	cout << "unset key" << endl;
//	cout << "set size ratio " << range[1] / range[0] << endl;
	for( size_t i=0 ; i<100000000 ; ++i )
	{
		// cout << "p [-1:" << range[0] + 1 << "][-1:" << range[1] + 1 << "] '-' w vec \n";
//		cout << "p [-1:" << range[0] + 1 << "][-1:" << range[1] + 1 << "] '-' pt 7 \n";
//		for( size_t i=0 ; i<n_solvent ; ++i )
//			cout << solvent1[i].coor << "\t" << solvent1[i].vel << "\n";
//		cout << "e" << endl;

//		double a = 1.001 * double(i);
//		for( size_t i=0 ; i<10000000 ; ++i ) a *= 1.0001;
//		clog << a << endl;

		vel_dist.add( solvent3 );

		test_poiseuille( solvent3 , force3 , n_solvent , n_virtual_particles , bc , indexer , indices3 , permutated_indices3 , begins3 , ends3 , rng3 , t3 , dt , mass , 4 );

		if( i  && ( !(i%100)))
		{

			clog << i << endl;
			ofstream fout( "dat/vel_dist.dat" );
			vel_dist.write( fout );
		}

	}


//	mass_point_vector_type2 solvent3_host = solvent3;
//
//	for( size_t i=0 ; i<indexer.number_of_cells() ; ++i )
//	{
//		cout << i << "\t";
//		cout << begins1[i] << "\t" << ends1[i] << "\t" << solvent1[i].coor << "\t" << solvent1[i].vel << "\t";
//		cout << begins2[i] << "\t" << ends2[i] << "\t" << solvent2[i].coor << "\t" << solvent2[i].vel  << "\t";
//		cout << begins3[i] << "\t" << ends3[i] << "\t" << solvent3_host[i].coor << "\t" << solvent3_host[i].vel  << "\n";
//	}

	return 0;
}

