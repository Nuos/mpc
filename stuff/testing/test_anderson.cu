/*
 * test_anderson.cu
 *
 *  Created on: Dec 15, 2010
 *      Author: karsten
 */


/*
 * streaming.cpp
 *
 *  Created on: Dec 6, 2010
 *      Author: karsten
 */

#include <iostream>
#include <vector>

#include <boost/static_assert.hpp>
#include <boost/type_traits.hpp>

// thrust includes
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/binary_search.h>

// cuda include
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// algebra includes
#include <boost/numeric/odeint/algebra/range_algebra.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_algebra.hpp>


#include <mpc/defines.hpp>
#include <mpc/point.hpp>
#include <mpc/mass_point.hpp>
#include <mpc/indexer.hpp>
#include <mpc/maxwell_velocity.hpp>
#include <mpc/collision.hpp>
#include <mpc/drand48_generator.hpp>

using namespace std;
using namespace mpc2;

using boost::numeric::odeint::range_algebra;
using boost::numeric::odeint::thrust_algebra;


const static size_t dim = 2;
typedef double value_type;
typedef point< value_type , dim > point_type;
typedef mass_point< point_type > mass_point_type;

typedef std::vector< mass_point_type > mass_point_vector_type1;
typedef std::vector< point_type > point_vector_type1;
typedef std::vector< size_t > cell_type1;
typedef std::vector< point< size_t , dim > > index_type1;
typedef std::vector< hiprandState > rng_state_type1;

typedef thrust::host_vector< mass_point_type > mass_point_vector_type2;
typedef thrust::host_vector< point_type > point_vector_type2;
typedef thrust::host_vector< size_t > cell_type2;
typedef thrust::host_vector< point< size_t , dim > > index_type2;
typedef thrust::host_vector< hiprandState > rng_state_type2;

typedef thrust::device_vector< mass_point_type > mass_point_vector_type3;
typedef thrust::device_vector< point_type > point_vector_type3;
typedef thrust::device_vector< size_t > cell_type3;
typedef thrust::device_vector< point< size_t , dim > > index_type3;
typedef thrust::device_vector< hiprandState > rng_state_type3;


const size_t n_solvent = 32;
const point_type range( 2.0 , 2.0 );
const point_type null_point( 0.0 , 0.0 );
const point_type spacing( 1.0 , 1.0 );
const value_type sqrt_kbT_mass = 1.0;



int main( int argc , char **argv )
{
	srand48( 325345777 );
	typedef drand48_generator rng_type;
	rng_type rng;


	typedef mpc2::regular_indexer< point_type ,
			with_start_point_tag ,
			check_and_throw_index_policy > regular_indexer_type;
	regular_indexer_type indexer( range , spacing );

	indexer.automatic_grid_shift( drand48 );
	clog << "Indexer start point : " << indexer.start() << endl;



	// initialize the solvent particles
	mass_point_vector_type1 solvent1;
	for( size_t i=0 ; i<n_solvent ; ++i )
	{
		mass_point_type mp;
		mp.coor[0] = drand48() * range[0];
		mp.coor[1] = drand48() * range[1];
		maxwell_velocity( mp.vel , sqrt_kbT_mass , rng );
		solvent1.push_back( mp );
	}

	mass_point_vector_type2 solvent2 = solvent1;
//	mass_point_vector_type3 solvent3 = solvent1;
	cell_type1 cell1( n_solvent );
	cell_type2 cell2( n_solvent );
//	cell_type3 cell3( n_solvent );

	cell_type1 begins1( indexer.number_of_cells() ) , ends1( indexer.number_of_cells() );
	cell_type2 begins2( indexer.number_of_cells() ) , ends2( indexer.number_of_cells() );
//	cell_type3 begins3( indexer.number_of_cells() ) , ends3( indexer.number_of_cells() );

	cout << 1 << endl;

	rng_state_type1 rng1( indexer.number_of_cells() );
	for( size_t i=0 ; i<indexer.number_of_cells() ; ++i ) hiprand_init( 1856234 , 0 , 0 , &rng1[i] );
	rng_state_type2 rng2 = rng1;
//	rng_state_type3 rng3 = rng1;


//	anderson( solvent1 , begins1 , ends1 , rng1 , indexer );
//	anderson( solvent2 , begins2 , ends2 , rng2 , indexer );
//	anderson( solvent3 , begins3 , ends3 , rng3 , indexer );
//
//	mass_point_vector_type2 solvent3_host = solvent3;
//
//	for( size_t i=0 ; i<indexer.number_of_cells() ; ++i )
//	{
//		cout << i << "\t";
//		cout << begins1[i] << "\t" << ends1[i] << "\t" << solvent1[i].coor << "\t" << solvent1[i].vel << "\t";
//		cout << begins2[i] << "\t" << ends2[i] << "\t" << solvent2[i].coor << "\t" << solvent2[i].vel  << "\t";
//		cout << begins3[i] << "\t" << ends3[i] << "\t" << solvent3_host[i].coor << "\t" << solvent3_host[i].vel  << "\n";
//	}

	cout << "set term x11" << endl;
	cout << "unset key" << endl;
	cout << "set size ratio " << range[1] / range[0] << endl;
	for( size_t i=0 ; i<100000 ; ++i )
	{
		cout << "p [-1:" << range[0] + 1 << "][-1:" << range[1] + 1 << "] '-' w vec \n";
		for( size_t i=0 ; i<n_solvent ; ++i )
			cout << solvent1[i].coor << "\t" << solvent1[i].vel << "\n";
		cout << "e" << endl;

		for( size_t i=0 ; i<rng1.size() ; ++i )
		{
			clog << "rng " << i << " " << rng1[i].d << " ";
			clog << rng1[i].v[0] << " " << rng1[i].v[1] << " " << rng1[i].v[2] << " " << rng1[i].v[3] << " " << rng1[i].v[4] << endl;
		}


		anderson( solvent1 , begins1 , ends1 , rng1 , indexer );
		getchar();
	}

	return 0;
}

