/*
 * test_anderson.cu
 *
 *  Created on: Dec 15, 2010
 *      Author: karsten
 */


/*
 * streaming.cpp
 *
 *  Created on: Dec 6, 2010
 *      Author: karsten
 */

#include <iostream>
#include <vector>

// cuda include
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// thrust includes
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/binary_search.h>
#include <thrust/random.h>

#include <mpc/defines.hpp>

using namespace std;


struct gaussian
{
	template< class Tuple >
	FUNC_DECL
	void operator()( Tuple t )
	{
		hiprandState state = thrust::get< 0 >( t );
		thrust::get< 1 >( t ) = hiprand_normal_double( &state );
	}
};

int main( int argc , char **argv )
{
	typedef thrust::random::default_random_engine::result_type result_type;
	thrust::random::default_random_engine rng1( 10 ) , rng2( 10 );


//	result_type r10 = rng1() , r20 = rng2();
//	for( size_t i=0 ; i<10 ; ++i )
//	{
//		result_type r1 = rng1() , r2 = rng2();
//		cout << i << "\t" << r1 << "\t" << r2 << "\n";
//		if( i == 2 ) rng2.seed( r10 );
//		if( i == 5 ) rng2.seed( r1 );
//	}
//
//	hiprandState state;
//	hiprand_init( 123 , 1 , 1 , &state );
//
//	for( size_t i=0 ; i<10 ; ++i )
//		cout << hiprand( &state ) << "\n";

	const size_t n = 32;
	typedef std::vector< hiprandState > seed_vector_type1;
	typedef thrust::host_vector< hiprandState > seed_vector_type2;
	typedef thrust::device_vector< hiprandState > seed_vector_type3;

	typedef std::vector< double > value_vector_type1;
	typedef thrust::host_vector< double > value_vector_type2;
	typedef thrust::device_vector< double > value_vector_type3;

	seed_vector_type1 seeds1( n );
	for( size_t i=0 ; i<n ; ++i ) hiprand_init( 1856234 , 3 * i , 1 , &seeds1[i] );
	seed_vector_type2 seeds2 = seeds1;
	seed_vector_type3 seeds3 = seeds1;

	value_vector_type1 values1( n );
	value_vector_type2 values2( n );
	value_vector_type3 values3( n );


	thrust::for_each(
			thrust::make_zip_iterator( thrust::make_tuple( seeds1.begin() , values1.begin() ) ) ,
			thrust::make_zip_iterator( thrust::make_tuple( seeds1.end() , values1.end() ) ) ,
			gaussian() );

	thrust::for_each(
			thrust::make_zip_iterator( thrust::make_tuple( seeds2.begin() , values2.begin() ) ) ,
			thrust::make_zip_iterator( thrust::make_tuple( seeds2.end() , values2.end() ) ) ,
			gaussian() );


	thrust::for_each(
			thrust::make_zip_iterator( thrust::make_tuple( seeds3.begin() , values3.begin() ) ) ,
			thrust::make_zip_iterator( thrust::make_tuple( seeds3.end() , values3.end() ) ) ,
			gaussian() );

	value_vector_type2 values3_host = values3;
	for( size_t i=0 ; i<n ; ++i )
		cout << values1[i] << "\t" << values2[i] << "\t" << values3_host[i] << "\n";



	return 0;
}

